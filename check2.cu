#include"check.h"
#include<hip/hip_runtime.h>
 void checkruntime(hipError_t err){
	if(err!=hipSuccess){
		printf("cuda runtime API error %d:%sd",(int)err,
				hipGetErrorString(err));
		exit(-1);
	}
}
void checkkernel(){
	hipError_t err=hipGetLastError();
	if(err!=hipSuccess){
		printf("cuda kernel launch error %d:%s\n",(int)err,hipGetErrorString(err));
		exit(-1);
	}
}
void checkfft(hipfftResult err){
	if(err!=HIPFFT_SUCCESS){
		printf("cuda fft API error:%d\n",(int)err);
		exit(-1);

	}
}
void checkcusolverDn(hipsolverStatus_t err){
	if(err!=HIPSOLVER_STATUS_SUCCESS){
		printf("cuda cusolverDn API error:%d\n",(int)err);
		exit(-1);
	}
}
void checkcusparse(hipsparseStatus_t err){
	if(err!=HIPSPARSE_STATUS_SUCCESS){
		printf("cuda cusparse API error:%d\n",(int)err);
		exit(-1);
	}
}
