
#include<hip/hip_runtime.h>
#include"gettime.c"
#include"tensor.c"
#include<assert.h>
#include"fft.h"
#include"svd.h"
#include"check.h"
int main(int arc,char** argc){
	int m,n,k,l;
	printf("input the size of tensor(m*n*k*l):");
	scanf("%d%d%d%d",&m,&n,&k,&l);
	printf("m=%d,n=%d,k=%d,l=%d\n",m,n,k,l);
	float* T=(float*)malloc(m*n*k*l*sizeof(float));
	float* S=(float*)malloc(sizeof(float)*(m<n)?m:n);
	hipComplex* U=(hipComplex*)malloc(sizeof(hipComplex)*m*((m<n)?m:n));
	hipComplex* V=(hipComplex*)malloc(sizeof(hipComplex)*n*((m<n)?m:n));
	hipfftComplex* A=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* B=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* C=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* D=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* E=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	if(!T||!A||!C||!D||!S||!U||!V){
		printf("host memory allocation failed");
		exit(-1);
	}
	double time1,time2,time3;
	time1=gettime();
	T=createtensor(m,n,k,l,T);
	printtensor(m,n,k,l,T);
	A=createctensor(m,n,k,l,A);
    printctensor(m,n,k,l,A);
    C=tensor_scalar(m,n,k,l,A,C);
    printctensor(m,n,k,l,C);
   svd(m*n,k*l,(hipComplex*)C,U,V,S);
    D=tensor_scalar_transpose(m,n,k,l,C,D);
    printctensor(m,n,k,l,D);
    E=tensor_scalartotensor(m,n,k,l,C,E);
    printctensor(m,n,k,l,E);
    fft(m,n,k,l,A,B);
    printctensor(m,n,k,l,B);
	time2=gettime();
	time3=time2-time1;
	printf("time:%.6f",time3);
    checkkernel();
	free(A);
	free(B);
	free(C);
	free(D);
	free(E);
	free(S);
	free(U);
	free(V);
	return 0;
}
