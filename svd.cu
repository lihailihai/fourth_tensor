#include"svd.h"
#include<hipsolver.h>
void svd(int m,int n,hipComplex* t,hipComplex* U,hipComplex* V,float* S){
     hipsolverHandle_t handle;
     hipsolverGesvdjInfo_t params=NULL;
     int* info=NULL;
     int echo=0;
     int lda=m;
     int ldu=m;
     int ldv=n;
     int lwork=0;
     hipComplex* work=NULL;
     float* s;
     hipComplex* u;
     hipComplex* v;
     hipsolverStatus_t status=HIPSOLVER_STATUS_SUCCESS;
     status=hipsolverDnCreate(&handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnCreateGesvdjInfo(&params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     hipError_t stat1=hipSuccess;
     hipError_t stat2=hipSuccess;
     hipError_t stat3=hipSuccess;
     hipError_t stat4=hipSuccess;
     stat1=hipMalloc((void**)&info,sizeof(int));
     stat2=hipMalloc((void**)&u,sizeof(hipComplex)*m*m);
     stat3=hipMalloc((void**)&v,sizeof(hipComplex)*n*n);
     stat4=hipMalloc((void**)&s,sizeof(float)*((m<n)?m:n));
     if(
    		 stat1!=hipSuccess||
    		 stat2!=hipSuccess||
    		 stat3!=hipSuccess||
    		 stat4!=hipSuccess){
    	 printf("cuda malloc error\n");
    	 exit(-1);
     }
     if(hipsolverDnCgesvdj_bufferSize(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 m,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 &lwork,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj_bufferSize failed\n");
    	 exit(-1);

     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("synchronize failed");
    	 exit(-1);
     }
     stat1=hipMalloc((void**)&work,sizeof(hipComplex)*lwork);
     assert(stat1==hipSuccess);
     if(hipsolverDnCgesvdj(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 lda,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 work,
    		 lwork,
    		 info,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj err\n");
    	 return;
     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("cuda synchronize err\n");
    	 return;
     }
     stat1=hipMemcpy(U,u,sizeof(hipComplex)*ldu*m,hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(V,v,sizeof(hipComplex)*ldv*n,hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(S,s,sizeof(float)*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     status=hipsolverDnDestroy(handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnDestroyGesvdjInfo(params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     stat1=hipFree(u);
     assert(stat1==hipSuccess);
     stat1=hipFree(v);
     assert(stat1==hipSuccess);
     stat1=hipFree(s);
     assert(stat1==hipSuccess);
}
