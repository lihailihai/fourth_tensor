#include"fft.h"
void fft_batch(int m,int n,int k,int l,hipComplex* t,hipComplex* ft)
{
	hipfftHandle plan;
	int rank=2;
	int N[rank-1]={l,k};
	int inembed[rank-1]={k,l};
	int onembed[rank-1]={k,l};
	int istride=1;
	int ostride=1;
	int idist=k*l;
	int odist=k*l;
	hipfftType type=HIPFFT_C2C;
	int batch=m*n;
	hipfftResult_t cufftstat1=HIPFFT_SUCCESS;
	hipfftResult_t cufftstat2=HIPFFT_SUCCESS;
	hipfftResult_t cufftstat3=HIPFFT_SUCCESS;
	hipError_t stat1=hipSuccess;
	hipError_t stat2=hipSuccess;
	hipError_t stat3=hipSuccess;
	hipError_t stat4=hipSuccess;
	hipError_t stat5=hipSuccess;
	cufftstat1=hipfftPlanMany(
			&plan,
			rank,
			N,
			inembed,
			istride,
			idist,
			onembed,
			ostride,
			odist,
			type,
			batch
			);
    stat1=hipDeviceSynchronize();
    hipfftComplex* idata;hipfftComplex* odata;
    stat2=hipMalloc((void**)&idata,sizeof(hipfftComplex)*m*n*k*l);
    stat3=hipMalloc((void**)&odata,sizeof(hipfftComplex)*m*n*k*l);
    stat4=hipMemcpy(
    		idata,
    		t,
    		sizeof(hipfftComplex)*m*n*k*l,
    		hipMemcpyHostToDevice);
    int direction=HIPFFT_FORWARD;
    cufftstat2=hipfftExecC2C(
            plan,
    		idata,
    		odata,
    		direction);
    stat5=hipMemcpy(
    		ft,
    		odata,
    		sizeof(hipfftComplex)*n*m*k*l,
    		hipMemcpyDeviceToHost);
    cufftstat3=hipfftDestroy(plan);
    if(
    		stat1!=hipSuccess||
    		stat2!=hipSuccess||
    		stat3!=hipSuccess||
    		stat4!=hipSuccess||
    		stat5!=hipSuccess){
    	printf("cuda runtime API error");
    	return;
    }
    if(
    		cufftstat1!=HIPFFT_SUCCESS||
    		cufftstat2!=HIPFFT_SUCCESS||
    		cufftstat3!=HIPFFT_SUCCESS){
    	printf("cufft API error");
    	return;
    }

    stat1=hipFree(idata);
    assert(stat1==hipSuccess);
    stat1=hipFree(odata);
    assert(stat1==hipSuccess);

}
void ifft_batch(int m,int n,int k,int l,hipComplex* t,hipComplex* ft)
{
	hipfftHandle plan;int rank=2;
	int N[rank-1]={l,k};
	int inembed[rank-1]={k,l};int onembed[rank-1]={k,l};
	int istride=1;int ostride=1;
	int idist=k*l; int odist=k*l;
	hipfftType type=HIPFFT_C2C;
	int batch=m*n;
	    hipfftResult_t cufftstat1=HIPFFT_SUCCESS;
		hipfftResult_t cufftstat2=HIPFFT_SUCCESS;
		hipfftResult_t cufftstat3=HIPFFT_SUCCESS;
		hipError_t stat1=hipSuccess;
		hipError_t stat2=hipSuccess;
		hipError_t stat3=hipSuccess;
		hipError_t stat4=hipSuccess;
		hipError_t stat5=hipSuccess;
	cufftstat1=hipfftPlanMany(
			&plan,
			rank,
			N,
			inembed,
			istride,
			idist,
			onembed,
			ostride,
			odist,
			type,
			batch
			);
    stat1=hipDeviceSynchronize();
    hipfftComplex* idata;hipfftComplex* odata;
    stat2=hipMalloc((void**)&idata,sizeof(hipfftComplex)*m*n*k*l);
    stat3=hipMalloc((void**)&odata,sizeof(hipfftComplex)*m*n*k*l);
    stat4=hipMemcpy(
    		idata,
    		t,
    		sizeof(hipfftComplex)*m*n*k*l
    		,hipMemcpyHostToDevice);
    int direction=HIPFFT_BACKWARD;
    cufftstat2=hipfftExecC2C(
            plan,
    		idata,
    		odata,
    		direction);
    stat5=hipMemcpy(
    		ft,
    		odata,
    		sizeof(hipfftComplex)*n*m*k*l,
    		hipMemcpyDeviceToHost);
    cufftstat3=hipfftDestroy(plan);
    assert(stat1==hipSuccess);
    assert(stat2==hipSuccess);
    assert(stat3==hipSuccess);
    assert(stat4==hipSuccess);
    assert(stat5==hipSuccess);
    if(
    		cufftstat1!=HIPFFT_SUCCESS||
    		cufftstat2!=HIPFFT_SUCCESS||
    		cufftstat3!=HIPFFT_SUCCESS){
    	printf("cufft API error");
    	exit(-1);

    }
    stat1=hipFree(idata);
    assert(stat1==hipSuccess);
    stat1=hipFree(odata);
    assert(stat1==hipSuccess);

}
