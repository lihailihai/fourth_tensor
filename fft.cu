#include"fft.h"
void fft(int m,int n,int k,int l,hipfftComplex* t,hipfftComplex* ft){
	int a=m*n*l;//HIPFFT_BACKWARD OR HIPFFT_FORWARD
	hipfftComplex* odata;
	hipfftComplex* idata;
	hipError_t stat1=hipSuccess;
	hipError_t stat2=hipSuccess;
	hipError_t stat3=hipSuccess;
	hipError_t stat4=hipSuccess;
	hipfftResult cufftstat1=HIPFFT_SUCCESS;
	hipfftResult cufftstat2=HIPFFT_SUCCESS;
	hipfftResult cufftstat3=HIPFFT_SUCCESS;
	stat1=hipMalloc((void**)&odata,sizeof(hipfftComplex)*m*n*k*l);
	stat2=hipMalloc((void**)&idata,sizeof(hipfftComplex)*m*n*k*l);
	assert(stat1==hipSuccess);
	assert(stat2==hipSuccess);
	stat3=hipMemcpy(
			idata,
			t,
			sizeof(hipfftComplex)*m*n*k*l
			,hipMemcpyHostToDevice);
	assert(hipSuccess==stat3);
	hipfftHandle plan;
	cufftstat1=hipfftPlan2d(&plan,a,k,HIPFFT_C2C);
    cufftstat2=hipfftExecC2C(
    		plan,
    		(hipfftComplex*)idata,
    		(hipfftComplex*)odata,
    		HIPFFT_FORWARD);
    hipDeviceSynchronize();
    stat4=hipMemcpy(
    		ft,
    		odata,
    		sizeof(hipfftComplex)*m*n*k*l,
    		hipMemcpyDeviceToHost);
    assert(stat4==hipSuccess);
    cufftstat3=hipfftDestroy(plan);
    if(cufftstat1!=HIPFFT_SUCCESS||
    		cufftstat2!=HIPFFT_SUCCESS||
    		cufftstat3!=HIPFFT_SUCCESS){
    	printf("cufft API error");
    	exit(-1);
    }
    hipFree(odata);
    hipFree(idata);
}
void ifft(int m ,int n,int k,int l,hipfftComplex* t,hipfftComplex* ft){
	int a=m*n*l;//HIPFFT_BACKWARD OR HIPFFT_FORWARD
		hipfftComplex* odata;
		hipfftComplex* idata;
		hipError_t stat1=hipSuccess;
		hipError_t stat2=hipSuccess;
		hipError_t stat3=hipSuccess;
		hipError_t stat4=hipSuccess;
		hipfftResult cufftstat1=HIPFFT_SUCCESS;
		hipfftResult cufftstat2=HIPFFT_SUCCESS;
		hipfftResult cufftstat3=HIPFFT_SUCCESS;
		stat1=hipMalloc((void**)&odata,sizeof(hipfftComplex)*m*n*k*l);
		stat2=hipMalloc((void**)&idata,sizeof(hipfftComplex)*m*n*k*l);
		stat3=hipMemcpy(
				idata,
				t,
				sizeof(hipfftComplex)*m*n*k*l,
				hipMemcpyHostToDevice);
		hipfftHandle plan;
		cufftstat1=hipfftPlan2d(&plan,a,k,HIPFFT_C2C);
	    cufftstat2=hipfftExecC2C(
	    		plan,
	    		(hipfftComplex*)idata,
	    		(hipfftComplex*)odata,
	    		HIPFFT_BACKWARD);
	    if(hipDeviceSynchronize()!=hipSuccess){
	    	printf("cuda synchronize failed");
	    	return;
	    }
	    stat4=hipMemcpy(
	    		ft,
	    		odata,
	    		sizeof(hipfftComplex)*m*n*k*l,
	    		hipMemcpyDeviceToHost);
	    cufftstat3=hipfftDestroy(plan);
	    assert(stat1==hipSuccess);
	    assert(stat2==hipSuccess);
	    assert(stat3==hipSuccess);
	    assert(stat4==hipSuccess);
	    assert(cufftstat1==HIPFFT_SUCCESS);
	    assert(cufftstat2==HIPFFT_SUCCESS);
	    assert(cufftstat3==HIPFFT_SUCCESS);
	    hipFree(odata);
	    hipFree(idata);
}
