#include "hip/hip_runtime.h"
#include"multi_mm.h"
void multi_mm(
		int* A_row_array,
		int* A_col_array,
		hipComplex* A_data_array,
		int* B_row_array,
		int* B_col_array,
		hipComplex* B_data_array,
		int m,
		int n,
		int min,
		int k,
		int l,
		hipsparseOperation_t transA,
		hipsparseOperation_t transB){
	int nnzA=0,nnzB=0,nnzC=0,baseC=0;
	int* C_row_array;
	int* C_col_array;
	hipComplex* C_data_array;
	hipComplex* d_A_data_array;
	hipComplex* d_B_data_array;
	hipComplex* d_C_data_array;
	int* d_A_row_array;
	int* d_A_col_array;
	int* d_B_row_array;
	int* d_B_col_array;
	int* d_C_row_array;
	int* d_C_col_array;
    hipError_t stat1=hipSuccess;
    hipError_t stat2=hipSuccess;
    hipError_t stat3=hipSuccess;
    hipError_t stat4=hipSuccess;
    hipError_t stat5=hipSuccess;
    hipError_t stat6=hipSuccess;
	stat1=hipMalloc((void**)&d_A_row_array,sizeof(int)*(m*k*l+1));
	stat2=hipMalloc((void**)&d_A_col_array,sizeof(int)*nnzA);
	stat3=hipMalloc((void**)&d_A_data_array,sizeof(hipComplex)*nnzA);
	stat4=hipMalloc((void**)&d_B_row_array,sizeof(int)*(n*k*l+1));
	stat5=hipMalloc((void**)&d_B_col_array,sizeof(int)*nnzB);
	stat6=hipMalloc((void**)&d_B_data_array,sizeof(hipComplex)*nnzB);
	if(
			stat1!=hipSuccess||
			stat2!=hipSuccess||
			stat3!=hipSuccess||
			stat4!=hipSuccess||
			stat5!=hipSuccess||
			stat6!=hipSuccess){
		printf("cuda malloc faild\n");
		return;
	}
	nnzA=d_A_row_array[m*k*l]-d_A_row_array[0];
	nnzB=d_B_row_array[n*k*l]-d_B_row_array[0];
	if(hipMemcpy(
			d_A_row_array,
			A_row_array,
			sizeof(int)*(m*l*k+1),
			hipMemcpyHostToDevice)!=hipSuccess){
		printf("cuda memcpy err 1\n");
		exit(-1);
	}
	if(hipMemcpy(
			d_A_col_array,
			A_col_array,
			sizeof(int)*nnzA,
			hipMemcpyHostToDevice)!=hipSuccess){
		printf("cuda memcpy err 2\n");
		exit(-1);
	}
	if(hipMemcpy(
			d_A_data_array,
			A_data_array,
			sizeof(hipComplex)*nnzA,
			hipMemcpyHostToDevice)!=hipSuccess){
		printf("cuda memcpy err 3\n");
		exit(-1);
	}
    if(hipMemcpy(
    		d_B_row_array,
    		B_row_array,
    		sizeof(int)*(n*k*l+1),
    		hipMemcpyHostToDevice)!=hipSuccess){
    	printf("cuda memcpy err 4\n");
    	exit(-1);
    }
    if(hipMemcpy(
    		d_B_col_array,
    		B_col_array,
    		sizeof(int)*nnzB,
    		hipMemcpyHostToDevice)!=hipSuccess){
    	printf("cuda memcpy err 5\n");
    	exit(-1);
    }
    if(hipMemcpy(
    		d_B_data_array,
    		B_data_array,
    		sizeof(hipComplex)*nnzB,
    		hipMemcpyHostToDevice)!=hipSuccess){
    	printf("cuda memcpy err 6\n");
    	exit(-1);
    }
    hipsparseHandle_t handle;
    if(hipsparseCreate(&handle)!=HIPSPARSE_STATUS_SUCCESS){
    	printf("cuaparsecreate handle failed\n");
    	return;
    }
    hipsparseMatDescr_t descrA;
    hipsparseMatDescr_t descrB;
    hipsparseMatDescr_t descrC;
    hipsparseStatus_t status=HIPSPARSE_STATUS_SUCCESS;
    status=hipsparseCreateMatDescr(&descrA);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseCreateMatDescr(&descrB);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseCreateMatDescr(&descrC);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    int* nnzTotalDevHostPtr=&nnzC;
    if(hipMalloc((void**)&d_C_row_array,sizeof(int)*(m*k*l+1))!=hipSuccess){
    	printf("cuda malloc error\n");
    	return;
    }
    if(hipsparseXcsrgemmNnz(
    		handle,
    		transA,
    		transB,
    		m*k*l,
    		n*k*l,
    		min*k*l,
    		descrA,
    		nnzA,
    		d_A_row_array,
    		d_A_col_array,
    		descrB,
    		nnzB,
    		d_B_row_array,
    		d_B_col_array,
    		descrC,
    		d_C_row_array,
    		nnzTotalDevHostPtr
    		)!=HIPSPARSE_STATUS_SUCCESS){
    	printf("gemmnz error\n");
    	exit(-1);
    }
    if(hipDeviceSynchronize()!=hipSuccess){
    	printf("synchronize error\n");
    	return;
    }
    if(NULL!=nnzTotalDevHostPtr){
    	nnzC=*nnzTotalDevHostPtr;
    }
    else{
    	hipMemcpy(
    			&nnzC,
    			d_C_row_array+m*k*l,
    			sizeof(int),
    			hipMemcpyDeviceToHost);
    	hipMemcpy(
    			&baseC,
    			d_C_row_array,
    			sizeof(int),
    			hipMemcpyDeviceToHost);
    	nnzC=-baseC;
    }
    C_row_array=(int*)malloc(sizeof(int)*(m*k*l+1));
    C_col_array=(int*)malloc(sizeof(int)*nnzC);
    C_data_array=(hipComplex*)malloc(sizeof(hipComplex)*nnzC);
    if(
    		!C_row_array||
    		!C_col_array||
    		!C_data_array){
    	printf("multi_mm malloc error");
    }
    hipError_t status2=hipSuccess;
    status2=hipMalloc((void**)&d_C_col_array,sizeof(int)*nnzC);
    assert(status2==hipSuccess);
    status2=hipMalloc((void**)&d_C_data_array,sizeof(hipComplex)*nnzC);
    assert(status2==hipSuccess);
    if(hipsparseCcsrgemm(
    		handle,
    		transA,
    		transB,
    		m*k*l,
    		n*k*l,
    		min*k*l,
    		descrA,
    		nnzA,
    		d_A_data_array,
    		d_A_row_array,
    		d_A_col_array,
    		descrB,
    		nnzB,
    		d_B_data_array,
    		d_B_row_array,
    		d_B_col_array,
    		descrC,
    		d_C_data_array,
    		d_C_row_array,
    		d_C_col_array
    		)!=HIPSPARSE_STATUS_SUCCESS){
    	printf("csrgemm error\n");
    	exit(-1);
    }
    status2=hipDeviceSynchronize();
    assert(status2==hipSuccess);
    status2=hipMemcpy(
    		C_row_array,
    		d_C_row_array,
    		sizeof(int)*(m*n*l+1),
    		hipMemcpyDeviceToHost);
    assert(status2==hipSuccess);
    status2=hipMemcpy(
    		C_col_array,
    		d_C_col_array,
    		sizeof(int)*nnzC,
    		hipMemcpyDeviceToHost);
    assert(status2==hipSuccess);
    status2=hipMemcpy(
    		C_data_array,
    		d_C_data_array,
    		sizeof(hipComplex)*nnzC,
    		hipMemcpyDeviceToHost);
    assert(status2==hipSuccess);
    status=hipsparseDestroyMatDescr(descrA);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseDestroyMatDescr(descrB);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseDestroyMatDescr(descrC);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status=hipsparseDestroy(handle);
    assert(status==HIPSPARSE_STATUS_SUCCESS);
    status2=hipFree(d_A_row_array);
    assert(status2==hipSuccess);
    status2=hipFree(d_A_col_array);
    assert(status2==hipSuccess);
    status2=hipFree(d_A_data_array);
    assert(status2==hipSuccess);
    status2=hipFree(d_B_row_array);
    assert(status2==hipSuccess);
    status2=hipFree(d_B_col_array);
    assert(status2==hipSuccess);
    status2=hipFree(d_B_data_array);
    assert(status2==hipSuccess);
}
