#include"check.h"
void checkruntime(hipError_t err){
	if(err!=hipSuccess){
		printf("cuda runtime API error %d:%sd",(int)err,
				hipGetErrorString(err));
		exit(-1);
	}
}
void checkkernel(){
	hipError_t err=hipGetLastError();
	if(err!=hipSuccess){
		printf("cuda kernel launch error %d:%s\n",(int)err,hipGetErrorString(err));
		exit(-1);
	}
}
void checkfft(hipfftResult err){
	if(err!=HIPFFT_SUCCESS){
		printf("cuda fft API error:%d",(int)err);
		exit(-1);

	}
}void checkcusolverDn(hipsolverStatus_t err){
	if(err!=HIPSOLVER_STATUS_SUCCESS){
		printf("cuda cusolverDn API error:%d",(int)err);
		exit(-1);
	}
}
